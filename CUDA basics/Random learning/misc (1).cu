
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if (id == 10) {
    	printf("ID %d\n", id);
    }
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 100000;

    std::cout << "Foo" << std::endl;
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
 
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    printf("A\n");
    hipMalloc(&d_a, bytes);
    printf("B\n");
    hipMalloc(&d_b, bytes);
    printf("C\n");
    hipMalloc(&d_c, bytes);
    printf("D\n");
 
    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sin(i)*sin(i);
        h_b[i] = cos(i)*cos(i);
    }
 
    // Copy host vectors to device
    printf("E\n");
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    printf("F\n");
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    printf("G\n");
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    printf("H\n");
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    printf("I\n");
 
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    printf("J\n");
 
    // Sum up vector c and print result divided by n, this should equal
    // 1 within error
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/(double)n);
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
